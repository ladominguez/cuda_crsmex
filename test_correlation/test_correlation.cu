#include "hip/hip_runtime.h"
//nvcc -arch=sm_30 -lcufft fft_batched.cu
#include <stdio.h>
#include <ctype.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <hip/hip_complex.h>
extern "C"{
#include <sacio.h>
#include <sac.h>
}


#define DATASIZE   1024
#define BATCH      2
#define MAX_ARRAY  1024
#define NSAC       2
#define MAX_PATH   100
#define N_FILENAME 2

#define GRID_DIMENSION  2
#define BLOCK_DIMENSION 8



/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void power_func( long int nelem, int npts,  hipfftComplex  *fft, hipfftComplex *power);
__global__ void correlation_coeff(hipfftComplex *power);

char *strstrip(char *s);


/********/
/* MAIN */
/********/
int main ()
{
    
    int       grid_size  = GRID_DIMENSION;
    int       block_size = BLOCK_DIMENSION;
    float     *data;
    float     beg, del;
    char      *line;
    FILE      *fid;
    size_t    len=0;
    int       count=0;
    int       nlen, nerr, max = MAX_ARRAY;
    char      infilename[]={"filenames.dat"} ;
    char      kname[ N_FILENAME ] ;
    float     yarray[MAX_ARRAY];
    

    dim3 DimGrid(grid_size, grid_size, grid_size);
    dim3 DimBlock(block_size, block_size, block_size);

    // --- Device side output data allocation
    hipfftComplex     *deviceOutputData; 
    hipfftComplex     *power;
    hipfftReal        *power_time;

    // --- Host side output data allocation
    int size_fft = DATASIZE / 2 + 1;
    hipfftComplex    *hostOutputData     = (   hipfftComplex*)malloc((size_fft) * BATCH * sizeof(hipfftComplex));
    hipfftComplex    *hostOutputPower    = (   hipfftComplex*)malloc((size_fft) * BATCH * sizeof(hipfftComplex)); 
    hipfftReal       *hostOutputPowerT   = (      hipfftReal*)malloc((DATASIZE) * BATCH * sizeof(hipfftReal)); 
    fprintf(stderr, "size_fft = %d\n",size_fft);

    // --- Host side input data allocation and initialization
    hipfftReal       *hostInputData = (hipfftReal*)malloc(DATASIZE*BATCH*sizeof(hipfftReal));
    data = (float *)malloc(NSAC * MAX_ARRAY * sizeof(float));

    // Lee nombres de archio
    fid = fopen(infilename,"r");
    if (fid == NULL){
        fprintf(stderr,"Couldn't open file %s\n",infilename);
        exit(-1);
    }

    // Lee archivos sac
    while (getline(&line, &len, fid) != -1)
    {
        line = strstrip(line);
        strcpy ( kname ,line ) ;
        rsac1( kname, yarray, &nlen, &beg, &del, &max, &nerr, strlen( kname ) ) ;
        if ( nerr != 0 ) {
                fprintf(stderr, "Error reading in SAC file: %s\n", kname);
                exit ( nerr ) ;
        }
        else {
                fprintf(stderr,"Reading SUCCESS: %s\n",kname);
                fprintf(stderr,"Number of samples read: %d\n\n",nlen);
        }
      
     	memcpy(&data[count*MAX_ARRAY], yarray, nlen*sizeof(float));
     	count++;
     }


    hipfftReal *deviceInputData; 
    gpuErrchk(hipMalloc((void**)&deviceInputData, DATASIZE * BATCH * sizeof(hipfftReal)));

    hipMemcpy(deviceInputData, data, DATASIZE * BATCH * sizeof(hipfftReal), hipMemcpyHostToDevice);



    gpuErrchk(hipMalloc((void**)&deviceOutputData, size_fft * BATCH * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc((void**)&power,            size_fft * BATCH * sizeof(hipfftComplex)));
    gpuErrchk(hipMalloc((void**)&power_time,       DATASIZE * BATCH * sizeof(hipfftReal)));

    // --- Batched 1D FFTs
    hipfftHandle handle_forward;
    hipfftHandle handle_inverse;
    int batch = BATCH;                          // --- Number of batched executions

    hipfftPlan1d( &handle_forward, DATASIZE, HIPFFT_R2C, batch);
    hipfftPlan1d( &handle_inverse, DATASIZE, HIPFFT_C2R, batch);

    // FFT
    hipfftExecR2C(handle_forward,  deviceInputData, deviceOutputData);

    power_func<<< DimGrid, DimBlock >>> (size_fft * BATCH, size_fft,  deviceOutputData, power );
    correlation_coeff <<< 4, 128 >>> (power); 

    int error_cufft=hipfftExecC2R(handle_inverse,  power, power_time);
    fprintf(stderr, "error_cufft = %d\n",error_cufft);

    // --- Device->Host copy of the results
    gpuErrchk(hipMemcpy(hostOutputData,     deviceOutputData, size_fft * BATCH * sizeof(hipfftComplex),    hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(hostOutputPower,    power,            size_fft * BATCH * sizeof(hipfftComplex),    hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(hostOutputPowerT,   power_time,       DATASIZE * BATCH * sizeof(hipfftReal),       hipMemcpyDeviceToHost));

    for (int i=0; i<BATCH; i++){
	printf("hostOutputPowerT[%d] = %f\n",i, hostOutputPowerT[DATASIZE*i]/(2));
    }


    hipfftDestroy(handle_forward);
    hipfftDestroy(handle_inverse);
    gpuErrchk(hipFree(deviceOutputData));
    gpuErrchk(hipFree(deviceInputData));
    gpuErrchk(hipFree(power));
    gpuErrchk(hipFree(power_time));
    hipDeviceSynchronize();
    hipDeviceReset();
    return EXIT_SUCCESS;

}

__global__ void correlation_coeff(hipfftComplex *power)
{
int bx = blockIdx.x;
int by = blockIdx.y;
int bz = blockIdx.z;

int thx = threadIdx.x;
int thy = threadIdx.y;
int thz = threadIdx.z;

int NumThread = blockDim.x*blockDim.y*blockDim.z;
int idThread  = (thx + thy*blockDim.x) + thz*(blockDim.x*blockDim.y);
int BlockId   =    (bx + by*gridDim.x) + bz*(gridDim.x*gridDim.y);

int uniqueid  = idThread + NumThread*BlockId;
// TODO
//printf("%d %d %d\n", uniqueid, idThread, BlockId);

}

__global__ void power_func(long int nelem, int npts, hipfftComplex *fft, hipfftComplex *power)
{
int bx = blockIdx.x;
int by = blockIdx.y;
int bz = blockIdx.z;

int thx = threadIdx.x;
int thy = threadIdx.y;
int thz = threadIdx.z;

int NumThread = blockDim.x*blockDim.y*blockDim.z;
int idThread  = (thx + thy*blockDim.x) + thz*(blockDim.x*blockDim.y);
int BlockId   =    (bx + by*gridDim.x) + bz*(gridDim.x*gridDim.y);

int uniqueid  = idThread + NumThread*BlockId;

if (uniqueid < nelem){
        power[uniqueid].x = (fft[uniqueid].x*fft[uniqueid].x + fft[uniqueid].y*fft[uniqueid].y)/npts;
	power[uniqueid].y = 0;
        //printf("Unique ID = %d - conj = %f\n",  uniqueid,  conj[uniqueid].y*-1);
}
}

char *strstrip(char *s)
{
        size_t size;
        char *end;

        size = strlen(s);

        if (!size)
                return s;

        end = s + size - 1;
        while (end >= s && isspace(*end))
                end--;
        *(end + 1) = '\0';

        while (*s && isspace(*s))
                s++;

        return s;
}
